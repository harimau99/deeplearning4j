/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/


//
// @author raver119@gmail.com
//

#include "testlayers.h"
#include <initializer_list>
#include <NDArrayFactory.h>
#include <ops/declarable/PlatformHelper.h>
#include <ops/declarable/CustomOperations.h>
#include <execution/Engine.h>

#ifdef HAVE_CUDNN

#include <ops/declarable/platform/cudnn/cudnnUtils.h>

#endif

using namespace nd4j;

class CuDnnTests : public testing::Test {
public:

};

static void printer(std::initializer_list<nd4j::ops::platforms::PlatformHelper*> helpers) {

    for (auto v:helpers) {
        nd4j_printf("Initialized [%s]\n", v->name().c_str());
    }
}


TEST_F(CuDnnTests, helpers_includer) {
    // we need this block, to make sure all helpers are still available within binary, and not optimized out by linker
#ifdef HAVE_CUDNN
    nd4j::ops::platforms::PLATFORM_conv2d_ENGINE_CUDA conv2d;


    printer({&conv2d});
#endif
}


TEST_F(CuDnnTests, mixed_helpers_test_1) {
#if defined(HAVE_CUDNN) && defined (HAVE_MKLDNN)
    nd4j_printf("Mixed platforms test\n", "");


    int bS=2, iH=4,iW=3,  iC=4,oC=3,  kH=3,kW=2,  sH=1,sW=1,  pH=0,pW=0,  dH=1,dW=1;
    int       oH=2,oW=2;
    int paddingMode = 0;             // 1-SAME, 0-VALID;
    int dataFormat  = 0;             // 1-NHWC, 0-NCHW

    auto input    = NDArrayFactory::create<float>('c', {bS, iC, iH, iW});
    auto weights  = NDArrayFactory::create<float>('c', {oC, iC, kH, kW});
    auto bias     = NDArrayFactory::create<float>('c', {oC}, {1,2,3});

    auto expOutput = NDArrayFactory::create<float>('c', {bS, oC, oH, oW}, {61.f,   61.f,  61.f,   61.f, 177.2f,  177.2f, 177.2f,  177.2f, 293.4f,  293.4f, 293.4f,  293.4f,  61.f,   61.f,  61.f,   61.f, 177.2f,  177.2f, 177.2f,  177.2f, 293.4f,  293.4f, 293.4f,  293.4f});
    auto zCUDA = expOutput.like();
    auto zMKL = expOutput.like();

    input = 2.;
    weights.linspace(0.1, 0.1);
    weights.permutei({2,3,1,0});

    input.syncToHost();
    weights.syncToHost();
    bias.syncToHost();

    nd4j::ops::conv2d op;

    // cuDNN part
    Context cuda(1);
    cuda.setTargetEngine(samediff::Engine::ENGINE_CUDA);
    cuda.setInputArray(0, &input);
    cuda.setInputArray(1, &weights);
    cuda.setInputArray(2, &bias);
    cuda.setOutputArray(0, &zCUDA);
    cuda.setIArguments({kH,kW,  sH,sW,  pH,pW,  dH,dW, paddingMode, dataFormat});
    auto statusCUDA = op.execute(&cuda);

    ASSERT_EQ(Status::OK(), statusCUDA);
    ASSERT_EQ(expOutput, zCUDA);

    // MKL-DNN part
    Context mkl(1);
    mkl.setTargetEngine(samediff::Engine::ENGINE_CPU);
    mkl.setInputArray(0, &input);
    mkl.setInputArray(1, &weights);
    mkl.setInputArray(2, &bias);
    mkl.setOutputArray(0, &zMKL);
    mkl.setIArguments({kH,kW,  sH,sW,  pH,pW,  dH,dW, paddingMode, dataFormat});
    auto statusMKL = op.execute(&mkl);

    zMKL.tickWriteHost();

    ASSERT_EQ(Status::OK(), statusMKL);
    ASSERT_EQ(expOutput, zMKL);
#endif
}