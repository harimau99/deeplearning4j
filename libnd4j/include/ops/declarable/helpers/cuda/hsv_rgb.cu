#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/
#include <ops/declarable/helpers/color_models_conv.h>
#include <ops/declarable/helpers/adjust_hue.h>
#include <ops/declarable/helpers/adjust_saturation.h>
#include <helpers/ConstantTadHelper.h>
#include <PointersManager.h>

namespace nd4j {
    namespace ops {
        namespace helpers {

            template <typename T>
            static void _CUDA_G rgbToHsvCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                              void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                              const Nd4jLong numOfTads, const int dimC) {

                const T* x = reinterpret_cast<const T*>(vx);
                T* z = reinterpret_cast<T*>(vz);

                __shared__ int rank;
                __shared__ Nd4jLong xDimCstride, zDimCstride;

                if (threadIdx.x == 0) {
                    rank = shape::rank(xShapeInfo);
                    xDimCstride = shape::stride(xShapeInfo)[dimC];
                    zDimCstride = shape::stride(zShapeInfo)[dimC];
                }
                __syncthreads();

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
                    const T* xTad = x + xTadOffsets[i];
                    T* zTad = z + zTadOffsets[i];

                    rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
                }
            }

            template <typename T>
            static void _CUDA_G hsvToRgbCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                             void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                             const Nd4jLong numOfTads, const int dimC) {

                const T* x = reinterpret_cast<const T*>(vx);
                T* z = reinterpret_cast<T*>(vz);

                __shared__ int rank;
                __shared__ Nd4jLong xDimCstride, zDimCstride;

                if (threadIdx.x == 0) {
                    rank = shape::rank(xShapeInfo);
                    xDimCstride = shape::stride(xShapeInfo)[dimC];
                    zDimCstride = shape::stride(zShapeInfo)[dimC];
                }
                __syncthreads();

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
                    const T* xTad = x + xTadOffsets[i];
                    T* zTad = z + zTadOffsets[i];

                    hsvToRgb<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
                }
            }

            ///////////////////////////////////////////////////////////////////
            template<typename T>
            static _CUDA_H void hsvToRgbCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                      const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                                      void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                                      const Nd4jLong numOfTads, const int dimC) {

                hsvToRgbCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
            }

            template<typename T>
            static _CUDA_H void rgbToHsvCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                     const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                                     void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                                     const Nd4jLong numOfTads, const int dimC) {

                rgbToHsvCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
            }


            void transform_hsv_rgb(nd4j::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
                auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

                const Nd4jLong numOfTads = packX.numberOfTads();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

                PointersManager manager(context, "hsv_to_rgb");

                NDArray::prepareSpecialUse({output}, {input});
                BUILD_SINGLE_SELECTOR(input->dataType(), hsvToRgbCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
                NDArray::registerSpecialUse({output}, {input});

                manager.synchronize();
            }

            void transform_rgb_hsv(nd4j::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
                auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

                const Nd4jLong numOfTads = packX.numberOfTads();

                const int threadsPerBlock = MAX_NUM_THREADS / 2;
                const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

                PointersManager manager(context, "rgb_to_hsv");

                NDArray::prepareSpecialUse({output}, {input});
                BUILD_SINGLE_SELECTOR(input->dataType(), rgbToHsvCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
                NDArray::registerSpecialUse({output}, {input});

                manager.synchronize();
            }
        }
    }
}