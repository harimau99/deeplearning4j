#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/random.h>
//#include <NativeOps.h>
#include <vector>
#include <memory>
#include <graph/Context.h>
#include <helpers/RandomLauncher.h>
#include <ShapeUtils.h>
#include <NDArrayFactory.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {

    /*
     * fillGammaKernel - fill up output with gamma distributed values
     *
     *  uList - uniformly distributed values set
     *  uLength - length of uList
     *  alpha - alpha param
     *  beta - beta param
     *  output - distributed output.
     * */
    template <typename T>
    static __global__ void fillGammaKernel(T* uList, Nd4jLong uLength, T* alpha, Nd4jLong* alphaShape,
            T* beta, Nd4jLong* betaShape, T* output, Nd4jLong* outputShape) {
        // fill up
        __shared__ Nd4jLong aLength;
        if (threadIdx.x == 0) {
            aLength = shape::length(alphaShape);
        }
        __syncthreads();

        for (auto k = blockIdx.x; k < (int)uLength; k += gridDim.x) {
            auto pos = k * aLength;
            auto u = uList[k]; // this is a vector
            for (auto e = threadIdx.x; e < (int)aLength; e += blockDim.x) {
                auto aIndex = shape::getIndexOffset(e, alphaShape);
                auto bIndex = betaShape?shape::getIndexOffset(e, betaShape):-1LL;
                auto betaV = T(beta != nullptr ? beta[bIndex] * u : u);
                auto zIndex = shape::getIndexOffset(e + pos, outputShape);

                output[zIndex] = math::nd4j_igamma<T, T, T>(alpha[aIndex], betaV);
            }
        }
    }

    template <typename T>
    static void fillRandomGamma_(LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output) {
        // To fill up output need to broadcast alpha and beta to the same shape and in
        Nd4jLong* broadcasted = nullptr;
        if (beta != nullptr)
            ShapeUtils::evalBroadcastShapeInfo(*alpha, *beta, true, broadcasted, context->getWorkspace());
        else
            broadcasted = alpha->shapeInfo();
        auto step = shape::length(broadcasted);
        auto shift = output->lengthOf() / step;

        auto copyAlpha = alpha;
        auto copyBeta = beta;
        if (beta != nullptr) {
            NDArray alphaBroadcasted(broadcasted, alpha->dataType(), true, context);
            NDArray betaBroadcasted(broadcasted, beta->dataType(), true, context);

            copyAlpha = new NDArray(alphaBroadcasted.applyTrueBroadcast(BroadcastOpsTuple::Assign(), *alpha));
            copyBeta = new NDArray(betaBroadcasted.applyTrueBroadcast(BroadcastOpsTuple::Assign(), *beta));
            copyAlpha->tickWriteDevice(); copyBeta->tickWriteDevice();
        }

        auto stream = context->getCudaStream();
        NDArray uniform = NDArrayFactory::create<T>('c', {shift}, context);
        uniform.syncToDevice();
        // fill up uniform with given length
        RandomLauncher::fillUniform(context, rng, &uniform, 0., 1.);

        fillGammaKernel<T><<<128, 128, 256, *stream>>>(uniform.dataBuffer()->specialAsT<T>(), shift,
                copyAlpha->dataBuffer()->specialAsT<T>(), copyAlpha->specialShapeInfo(),
                beta?copyBeta->dataBuffer()->specialAsT<T>():(T*)nullptr,
                beta?copyBeta->specialShapeInfo():(Nd4jLong*)nullptr,
                output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());

        if (beta != nullptr) {
            delete copyAlpha;
            delete copyBeta;
            //delete broadcasted;
        }

    }

    void fillRandomGamma(LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output) {
        if (beta)
            NDArray::prepareSpecialUse({output}, {alpha, beta});
        else
            NDArray::prepareSpecialUse({output}, {alpha});
        BUILD_SINGLE_SELECTOR(output->dataType(), fillRandomGamma_, (context, rng, alpha, beta, output), FLOAT_NATIVE);
        if (beta)
            NDArray::registerSpecialUse({output}, {alpha, beta});
        else
            NDArray::prepareSpecialUse({output}, {alpha});
    }
    BUILD_SINGLE_TEMPLATE(template void fillRandomGamma_, (LaunchContext* context, graph::RandomGenerator& rng, NDArray* alpha, NDArray* beta, NDArray* output), FLOAT_NATIVE);


    /*
     * algorithm Poisson generator based upon the inversion by sequential search
     *
    init:
         Let x ← 0, p ← e−λ, s ← p.
         using uniformly random sequence U (u in U) distributed at [0, 1].
    while u > s do:
         x ← x + 1.
         p ← p * λ / x.
         s ← s + p.
    return x.
     * */
    template <typename T>
    static __global__ void fillPoissonKernel(T* uList, Nd4jLong uLength, T* lambda, Nd4jLong* lambdaShape, T* output,
            Nd4jLong* outputShape) {

        __shared__ Nd4jLong step;

        if (threadIdx.x == 0) {
            step = shape::length(lambdaShape);
        }
        __syncthreads();

        for (auto k = blockIdx.x; k < (int)uLength; k += gridDim.x) {
            auto pos = k * step;
            auto u = uList[k];
            for (auto e = threadIdx.x; e < step; e += blockDim.x) {
                auto p = math::nd4j_exp<T,T>(-lambda[e]);
                auto s = p;
                auto x = T(0.f);
                auto lIndex = shape::getIndexOffset(e, lambdaShape);
                auto zIndex = shape::getIndexOffset(e + pos, outputShape);
                while (u > s) {
                    x += T(1.);
                    p *= lambda[lIndex] / x;
                    s += p;
                }
                output[zIndex] = x;
            }
        }
    }

    template <typename T>
    static void fillRandomPoisson_(LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output) {
        auto shift = output->lengthOf() / lambda->lengthOf();
        NDArray uniform('c', {shift}, output->dataType());
        auto stream = context->getCudaStream();
        // fill up uniform with given length
        RandomLauncher::fillUniform(context, rng, &uniform, 0., 1.);
        fillPoissonKernel<T><<<128, 256, 128, *stream>>>(uniform.dataBuffer()->specialAsT<T>(), uniform.lengthOf(),
                lambda->dataBuffer()->specialAsT<T>(), lambda->specialShapeInfo(),
                output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());
    }

    void fillRandomPoisson(LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {lambda});
        BUILD_SINGLE_SELECTOR(output->dataType(), fillRandomPoisson_, (context, rng, lambda, output), FLOAT_NATIVE);
        NDArray::registerSpecialUse({output}, {lambda});
    }

    BUILD_SINGLE_TEMPLATE(template void fillRandomPoisson_, (LaunchContext* context, graph::RandomGenerator& rng, NDArray* lambda, NDArray* output), FLOAT_NATIVE);

    template <typename T>
    static __global__ void fillUniformKernel(graph::RandomGenerator* devRng, T from, T to, T* output, Nd4jLong* outputShape) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        __shared__ Nd4jLong outputLen;

        if (0 == threadIdx.x) {
            outputLen = shape::length(outputShape);
        }
        __syncthreads();

        for (auto i = start; i < outputLen; i += step) {
            auto zIndex = shape::getIndexOffset(i, outputShape);
            output[zIndex] = devRng->relativeT<T>(i, from, to);
        }

    }

    template <typename T>
    static void fillRandomUniform_(LaunchContext* context, graph::RandomGenerator& rng, NDArray* min, NDArray* max, NDArray* output) {
        T minVal = T(0);
        T maxVal = DataTypeUtils::infOrMax<T>();
        if (min)
            minVal = min->t<T>(0);
        if (max)
            maxVal = max->t<T>(0);

        if (output->isR())
            RandomLauncher::fillUniform(context, rng, output, minVal, maxVal);
        else {
            auto stream = context->getCudaStream();
            graph::RandomGenerator *devRng;
            auto err = hipMalloc(&devRng, sizeof(graph::RandomGenerator));
            if (err != 0) {
                cuda_exception::build("fillRandomUniform_: Cannot allocate device memory for random generator due error", err);
            }

            err = hipMemcpy(devRng, &rng, sizeof(graph::RandomGenerator), hipMemcpyHostToDevice);
            if (err != 0) {
                cuda_exception::build("fillRandomUniform_: Cannot copy random generator to device", err);
            }
            auto outputBuf = output->dataBuffer()->specialAsT<T>();
            auto outputShape = output->specialShapeInfo();
            fillUniformKernel<T><<<128, 128, 128, *stream>>>(devRng, minVal, maxVal, outputBuf, outputShape);

            err = hipStreamSynchronize(*stream);
            if (err != 0) {
                cuda_exception::build("fillRandomUniform_: Cannot successfully finish kernel call", err);
            }

            err = hipFree(devRng);
            if (err != 0) {
                cuda_exception::build("fillRandomUniform_: Cannot deallocate device memory for random generator", err);
            }
        }
    }

    void fillRandomUniform(LaunchContext* context, graph::RandomGenerator& rng, NDArray* min, NDArray* max, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), fillRandomUniform_, (context, rng, min, max, output), NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void fillRandomUniform_, (LaunchContext* context,
            graph::RandomGenerator& rng, NDArray* min, NDArray* max, NDArray* output), NUMERIC_TYPES);

}
}
}