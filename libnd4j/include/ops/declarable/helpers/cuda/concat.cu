#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(void* pVx,  void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis) {

    T* z = reinterpret_cast<T*>(vz);
    __shared__ Nd4jLong zLen, totalThreads;
    __shared__ int rank;

    if (threadIdx.x == 0) {
        zLen = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    Nd4jLong coords[MAX_RANK];

    for (uint64_t i = tid; i < zLen; i += totalThreads) {
        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        int inArrIdx = 0;
        Nd4jLong *xShapeInfo = reinterpret_cast<Nd4jLong **>(pxShapeInfo)[inArrIdx];

        while (coords[axis] >= xShapeInfo[axis + 1]) {
            coords[axis] -= xShapeInfo[axis + 1];
            xShapeInfo = reinterpret_cast<Nd4jLong **>(pxShapeInfo)[++inArrIdx];
        }

        const auto *x = reinterpret_cast<T *>(reinterpret_cast<void **>(pVx)[inArrIdx]);
        const auto xOffset = shape::getOffset(xShapeInfo, coords);

        z[zOffset] = x[xOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void concatCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                        void* pVx, void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis) {

    concatCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(pVx, pxShapeInfo, vz, zShapeInfo, axis);
}
BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, void* pVx, void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis), LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
void concat(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output, const int axis) {

    const int numOfArrs = inArrs.size();

    const bool isZcontin = output.strideAt(axis) == 1;

    bool areInputsContin    = inArrs[0]->strideAt(axis) == 1;
    bool allInputsSameOrder = true;

    if(isZcontin) {
        for (uint i = 1; i < inArrs.size(); ++i) {
            areInputsContin    &= (inArrs[i]->strideAt(axis) == 1);
            allInputsSameOrder &= inArrs[i-1]->ordering() == inArrs[i]->ordering();
            if(!areInputsContin || !allInputsSameOrder)
                break;
        }
    }

    const bool luckCase = isZcontin && areInputsContin && allInputsSameOrder && output.ordering() == inArrs[0]->ordering();

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->syncToDevice();
    output.syncToDevice();

    if(luckCase) {

        const auto sizeofT    = output.sizeOfT();
        const uint zDim       = output.sizeAt(axis);

        for (uint i = 0; i < output.lengthOf() / zDim; ++i) {

            const auto iShift = i * sizeofT;
            void* z = static_cast<int8_t*>(output.getSpecialBuffer()) + zDim * iShift;

            for (uint j = 0; j < numOfArrs; ++j) {
                const auto xDim = inArrs[j]->sizeAt(axis);
                void* x = static_cast<int8_t*>(inArrs[j]->getSpecialBuffer()) + xDim * iShift;
                const auto memSizeToCopy = xDim * sizeofT;
                hipMemcpyAsync(z, x, memSizeToCopy, hipMemcpyDeviceToDevice, *context->getCudaStream());
                z = static_cast<int8_t*>(z) + memSizeToCopy;
            }
        }

        if(hipStreamSynchronize(*context->getCudaStream()) != 0)
            throw std::runtime_error("concat cuda: luckCase failed!");
    }
    else {      // general (slower) case

        const int threadsPerBlock = 256;
        const int blocksPerGrid = 512;
        const int sharedMem = 512;

        // prepare arrays of pointers on buffers and shapes
        std::vector<void*> hInBuffers(numOfArrs);
        std::vector<Nd4jLong*> hInShapeInfo(numOfArrs);

        for(int i = 0; i < numOfArrs; ++i) {
            hInBuffers[i]   = inArrs[i]->getSpecialBuffer();
            hInShapeInfo[i] = inArrs[i]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "helpers::concat");

        void* dInBuffers   = manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
        void* dInShapeInfo = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));

        BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), dInBuffers, dInShapeInfo, output.specialBuffer(), output.specialShapeInfo(), axis), LIBND4J_TYPES);

        manager.synchronize();
    }

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->tickReadDevice();
    output.tickWriteDevice();
}

}
}
}