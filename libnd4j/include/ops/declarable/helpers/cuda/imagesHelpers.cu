#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <op_boilerplate.h>
#include <ops/declarable/helpers/imagesHelpers.h>
#include <helpers/ConstantTadHelper.h>
#include <ops/declarable/helpers/adjust_hue.h>
#include <PointersManager.h>


namespace nd4j    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
// for example xShapeInfo = {2,3,4}, zShapeInfo = {2,1,4}
template<typename T>
__global__ void rgbToGrsCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int dimC) {

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong zLen, *sharedMem;
	__shared__ int rank;	// xRank == zRank

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

		zLen = shape::length(zShapeInfo);
		rank = shape::rank(zShapeInfo);
	}
	__syncthreads();

	Nd4jLong* coords = sharedMem + threadIdx.x * rank;

	for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < zLen; i +=  gridDim.x * blockDim.x) {

		if (dimC == (rank - 1) && 'c' == shape::order(xShapeInfo) && 1 == shape::elementWiseStride(xShapeInfo) && 'c' == shape::order(zShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo)) {
			const auto xStep = i*3;
            z[i] = 0.2989f * x[xStep] + 0.5870f * x[xStep + 1] + 0.1140f * x[xStep + 2];
		}
		else {

	    	shape::index2coords(i, zShapeInfo, coords);

            const auto zOffset  = shape::getOffset(zShapeInfo, coords);
            const auto xOffset0 = shape::getOffset(xShapeInfo, coords);
            const auto xOffset1 = xOffset0 + shape::stride(xShapeInfo)[dimC];
            const auto xOffset2 = xOffset1 + shape::stride(xShapeInfo)[dimC];

            z[zOffset] = 0.2989f * x[xOffset0] + 0.5870f * x[xOffset1] + 0.1140f * x[xOffset2];
		}
	}
}

///////////////////////////////////////////////////////////////////
template<typename T>
linkage void rgbToGrsCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int dimC) {

	rgbToGrsCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, dimC);
}

///////////////////////////////////////////////////////////////////
void transformRgbGrs(nd4j::LaunchContext* context, const NDArray& input, NDArray& output, const int dimC) {

	PointersManager manager(context, "rgbToGrs");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = input.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

	NDArray::prepareSpecialUse({&output}, {&input});
	BUILD_SINGLE_SELECTOR(input.dataType(), rgbToGrsCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), dimC), NUMERIC_TYPES);
	NDArray::registerSpecialUse({&output}, {&input});

	manager.synchronize();
}


///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G rgbToHsvCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                  void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                  const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G hsvToRgbCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                 void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                 const Nd4jLong numOfTads, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
    T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {
        const T* xTad = x + xTadOffsets[i];
        T* zTad = z + zTadOffsets[i];

        hsvToRgb<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static _CUDA_H void hsvToRgbCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                          const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                          void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                          const Nd4jLong numOfTads, const int dimC) {

    hsvToRgbCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

template<typename T>
static _CUDA_H void rgbToHsvCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                         const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                         void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                         const Nd4jLong numOfTads, const int dimC) {

    rgbToHsvCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, dimC);
}

///////////////////////////////////////////////////////////////////
void transformHsvRgb(nd4j::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {

    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "hsv_to_rgb");

    NDArray::prepareSpecialUse({output}, {input});
    BUILD_SINGLE_SELECTOR(input->dataType(), hsvToRgbCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
void transformRgbHsv(nd4j::LaunchContext* context, const NDArray* input, NDArray* output, const int dimC) {
    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "rgb_to_hsv");

    NDArray::prepareSpecialUse({output}, {input});
    BUILD_SINGLE_SELECTOR(input->dataType(), rgbToHsvCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input});

    manager.synchronize();
}












}
}
}

