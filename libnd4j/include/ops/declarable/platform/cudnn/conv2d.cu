/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace nd4j {
    namespace ops {
        namespace platforms {
            PLATFORM_IMPL(conv2d, ENGINE_CUDA) {
                auto handle = reinterpret_cast<hipdnnHandle_t *>(block.launchContext()->getCuDnnHandle());
                auto res = hipdnnSetStream(*handle, *block.launchContext()->getCudaStream());
                if (res != 0)
                    throw nd4j::cuda_exception::build("Can't set stream for cuDNN", res);

                auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
                auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
                auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

                auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

                NDArray::prepareSpecialUse({output}, {input, weights, bias});

                int sH = INT_ARG(2);                                                        // strides height
                int sW = INT_ARG(3);                                                        // strides width
                int pH = INT_ARG(4);                                                        // paddings height
                int pW = INT_ARG(5);                                                        // paddings width
                int dH = INT_ARG(6);                                                        // dilations height
                int dW = INT_ARG(7);                                                        // dilations width
                int isSameMode = INT_ARG(8);                                                // 0-VALID, 1-SAME
                bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW,  1-NHWC

                int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
                int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

                int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
                int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
                ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);
                ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, isSameMode);

                auto dtype = cudnnDataType(input->dataType());


                hipdnnTensorDescriptor_t src;
                hipdnnCreateTensorDescriptor(&src);
                res = hipdnnSetTensor4dDescriptorEx(src, dtype, input->sizeAt(0), input->sizeAt(1), input->sizeAt(2), input->sizeAt(3), input->strideAt(0), input->strideAt(1), input->strideAt(2), input->strideAt(3));
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx src failed", res);

                // TODO: we definitely want NHWC here as well
                hipdnnFilterDescriptor_t wght;
                hipdnnCreateFilterDescriptor(&wght);
                res = hipdnnSetFilter4dDescriptor(wght, dtype, HIPDNN_TENSOR_NCHW, oC, iC, kH, kW);
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetFilter4dDescriptor failed", res);

                hipdnnConvolutionDescriptor_t cdc;
                hipdnnCreateConvolutionDescriptor(&cdc);
                res = hipdnnSetConvolution2dDescriptor(cdc, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, dtype);
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetConvolution2dDescriptor failed", res);

                hipdnnTensorDescriptor_t dst;
                hipdnnCreateTensorDescriptor(&dst);
                res = hipdnnSetTensor4dDescriptorEx(dst, dtype, output->sizeAt(0), output->sizeAt(1), output->sizeAt(2), output->sizeAt(3), output->strideAt(0), output->strideAt(1), output->strideAt(2), output->strideAt(3));
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx dst failed", res);

                // TODO: workspace algorithms are supposed to be faster, so we should use it here if we have enough memory
                hipdnnConvolutionFwdAlgo_t algo;
                res = hipdnnGetConvolutionForwardAlgorithm(*handle, src, wght, cdc, dst, HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &algo);
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnGetConvolutionForwardAlgorithm failed", res);

                // TODO: should be float if dtype is half/float, and double otherwise
                float alpha = 1.0f;
                float beta = 0.0f;
                res = hipdnnConvolutionForward(*handle, &alpha, src, input->specialBuffer(), wght, weights->specialBuffer(), cdc, algo, nullptr, 0, &beta, dst, output->specialBuffer());
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnConvolutionForward failed", res);


                if (bias != nullptr) {
                    hipdnnTensorDescriptor_t bs;
                    hipdnnCreateTensorDescriptor(&bs);
                    if (isNCHW) {
                        res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NCHW, dtype, 1, bias->lengthOf(), 1, 1);
                        if (res != 0)
                            throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
                    } else {
                        res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NHWC, dtype, 1, 1, 1, bias->lengthOf());
                        if (res != 0)
                            throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
                    }

                    res = hipdnnAddTensor(*handle, &alpha, bs, bias->specialBuffer(), &alpha, dst, output->specialBuffer());
                    if (res != 0)
                        throw nd4j::cuda_exception::build("hipdnnAddTensor failed", res);
                }


                NDArray::registerSpecialUse({output}, {input, weights, bias});

                return Status::OK();
            }

            PLATFORM_CHECK(conv2d, ENGINE_CUDA) {
                return true;
            }
        }
    }
}
