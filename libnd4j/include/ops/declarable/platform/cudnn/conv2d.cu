/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//


#include "cudnnUtils.h"

namespace nd4j {
    namespace ops {
        namespace platforms {
            PLATFORM_IMPL(conv2d, ENGINE_CUDA) {
                auto handle = reinterpret_cast<hipdnnHandle_t *>(block.launchContext()->getCuDnnHandle());
                auto res = hipdnnSetStream(*handle, *block.launchContext()->getCudaStream());
                if (res != 0)
                    throw nd4j::cuda_exception::build("Can't set stream for cuDNN", res);

                auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
                auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
                auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

                auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

                int sH = INT_ARG(2);                                                        // strides height
                int sW = INT_ARG(3);                                                        // strides width
                int pH = INT_ARG(4);                                                        // paddings height
                int pW = INT_ARG(5);                                                        // paddings width
                int dH = INT_ARG(6);                                                        // dilations height
                int dW = INT_ARG(7);                                                        // dilations width
                int isSameMode = INT_ARG(8);                                                // 0-VALID, 1-SAME
                bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW,  1-NHWC

                int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
                int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

                auto dtype = cudnnDataType(input->dataType());


                hipdnnTensorDescriptor_t src;
                hipdnnCreateTensorDescriptor(&src);
                res = hipdnnSetTensor4dDescriptorEx(src, dtype, input->sizeAt(0), input->sizeAt(1), input->sizeAt(2), input->sizeAt(3), input->strideAt(0), input->strideAt(1), input->strideAt(2), input->strideAt(3));
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx src failed", res);

                hipdnnFilterDescriptor_t wght;
                hipdnnCreateFilterDescriptor(&wght);
                res = hipdnnSetFilter4dDescriptor(wght, dtype, HIPDNN_TENSOR_NCHW, weights->sizeAt(3), weights->sizeAt(2), kH, kW);
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetFilter4dDescriptor failed", res);

                hipdnnConvolutionDescriptor_t cdc;
                hipdnnCreateConvolutionDescriptor(&cdc);
                res = hipdnnSetConvolution2dDescriptor(cdc, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, dtype);
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetConvolution2dDescriptor failed", res);

                hipdnnTensorDescriptor_t dst;
                hipdnnCreateTensorDescriptor(&dst);
                res = hipdnnSetTensor4dDescriptorEx(dst, dtype, output->sizeAt(0), output->sizeAt(1), output->sizeAt(2), output->sizeAt(3), output->strideAt(0), output->strideAt(1), output->strideAt(2), output->strideAt(3));
                if (res != 0)
                    throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx dst failed", res);

                return Status::OK();
            }

            PLATFORM_CHECK(conv2d, ENGINE_CUDA) {
                return true;
            }
        }
    }
}
