/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace nd4j      {
namespace ops       {
namespace platforms {


static void conv2dCUDNN(const LaunchContext* context,
                        const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output,
                        const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW,
                        const int paddingMode, const bool isNCHW) {

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);
    ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, paddingMode);

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    auto err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: can't set stream for cuDNN", err);

    hipdnnTensorFormat_t format = isNCHW ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;

    // input descriptor
    hipdnnTensorDescriptor_t x;
    hipdnnCreateTensorDescriptor(&x);
    if(input->ews() == 1)
        err = hipdnnSetTensor4dDescriptor(x, format, cudnnDataType(input->dataType()), bS, iC, iH, iW);
    else
        err = hipdnnSetTensor4dDescriptorEx(x, cudnnDataType(input->dataType()), bS, iC, iH, iW, input->strideAt(0), input->strideAt(indIOioC), input->strideAt(indIiH), input->strideAt(indIiH + 1));
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for input failed", err);

    // weights descriptor
    hipdnnFilterDescriptor_t w;
    hipdnnCreateFilterDescriptor(&w);
    err = hipdnnSetFilter4dDescriptor(w, cudnnDataType(weights->dataType()), format, oC, iC, kH, kW);
    if(err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnSetFilter4dDescriptor failed", err);

    // output descriptor
    hipdnnTensorDescriptor_t z;
    hipdnnCreateTensorDescriptor(&z);
    if(output->ews() == 1)
        err = hipdnnSetTensor4dDescriptor(z, format, cudnnDataType(output->dataType()), bS, oC, oH, oW);
    else
        err = hipdnnSetTensor4dDescriptorEx(z, cudnnDataType(output->dataType()), bS, oC, oH, oW, output->strideAt(0), output->strideAt(indIOioC), output->strideAt(indOoH), output->strideAt(indOoH + 1));
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor/hipdnnSetTensor4dDescriptorEx for output failed", err);

    // description of convolution
    hipdnnConvolutionDescriptor_t conv;
    hipdnnCreateConvolutionDescriptor(&conv);
    err = hipdnnSetConvolution2dDescriptor(conv, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, cudnnDataType(output->dataType()));
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnSetConvolution2dDescriptor failed", err);

    // algorithm description
    hipdnnConvolutionFwdAlgo_t algo;
    err = hipdnnGetConvolutionForwardAlgorithm(*handle, x, w, conv, z, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnGetConvolutionForwardAlgorithm failed", err);

    // allocate auxiliary device memory, abbreviation ws means workspace
    size_t wsSize;
    err = hipdnnGetConvolutionForwardWorkspaceSize(*handle, x, w, conv, z, algo, &wsSize);
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnGetConvolutionForwardWorkspaceSize failed", err);
    void* wsData;
    auto cudaErr = hipMalloc(&wsData, wsSize);
    if (cudaErr != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipMalloc for auxiliary workspace memory failed", cudaErr);

    float  alpha32(1), beta32(0);
    double alpha64(1), beta64(0);
    const void* alpha = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&alpha32) : reinterpret_cast<const void*>(&alpha64);
    const void* beta  = output->sizeOfT() <= 4 ? reinterpret_cast<const void*>(&beta32)  : reinterpret_cast<const void*>(&beta64);

    NDArray::prepareSpecialUse({output}, {input, weights, bias});

    // run calculation
    err = hipdnnConvolutionForward(*handle, alpha, x, input->getSpecialBuffer(), w, weights->getSpecialBuffer(), conv, algo, wsData, wsSize, beta, z, output->specialBuffer());
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnConvolutionForward failed", err);

    // add bias if it is present
    if (bias != nullptr) {
        hipdnnTensorDescriptor_t b;
        hipdnnCreateTensorDescriptor(&b);
        err = hipdnnSetTensor4dDescriptor(b, format, cudnnDataType(output->dataType()), 1, isNCHW ? bias->lengthOf() : 1, 1, isNCHW ? 1: bias->lengthOf());
        if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnSetTensor4dDescriptor for bias failed", err);
        err = hipdnnAddTensor(*handle, &alpha, b, bias->getSpecialBuffer(), &beta, z, output->specialBuffer());
        if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipdnnAddTensor failed", err);
    }

    cudaErr = hipFree(wsData);
    if (cudaErr != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: hipFree for auxiliary workspace memory failed", cudaErr);

    NDArray::registerSpecialUse({output}, {input, weights, bias});
}

PLATFORM_IMPL(conv2d, ENGINE_CUDA) {

    auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
    auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

    auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

    int sH = INT_ARG(2);                                                        // strides height
    int sW = INT_ARG(3);                                                        // strides width
    int pH = INT_ARG(4);                                                        // paddings height
    int pW = INT_ARG(5);                                                        // paddings width
    int dH = INT_ARG(6);                                                        // dilations height
    int dW = INT_ARG(7);                                                        // dilations width
    int isSameMode = INT_ARG(8);                                                // 0-VALID, 1-SAME
    bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW,  1-NHWC

    int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
    int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    std::string expectedWeightsShape = ShapeUtils::shapeAsString({kH, kW, iC, oC});
    REQUIRE_TRUE(expectedWeightsShape == ShapeUtils::shapeAsString(weights), 0, "CUSTOM CONV2D CUDNN OP: wrong shape of weights array, expected is %s, but got %s instead !", expectedWeightsShape.c_str(), ShapeUtils::shapeAsString(weights).c_str());
    if (bias) {
        REQUIRE_TRUE(bias->rankOf() <= 2 && oC == bias->lengthOf(), 0, "CUSTOM CONV2D CUDNN OP: wrong shape of array with biases, expected rank, length: <=2, %i, but got %i, %i instead !", oC, bias->rankOf(), bias->lengthOf());
        REQUIRE_TRUE((bias->rankOf() == 1 && bias->strideAt(0) == 1) || (bias->rankOf() == 2 && bias->sizeAt(0) == 1 && bias->strideAt(1) == 1) || (bias->rankOf() == 2 && bias->sizeAt(1) == 1 && bias->strideAt(0) == 1), 0, "CUSTOM CONV2D CUDNN OP: bias array should be contiguous in memory !");
    }

    conv2dCUDNN(block.launchContext(), input, weights, bias, output, kH,kW,sH,sW,pH,pW,dH,dW, isSameMode, isNCHW);

    return Status::OK();
}

// PLATFORM_IMPL(conv2d, ENGINE_CUDA) {

//     auto handle = reinterpret_cast<hipdnnHandle_t *>(block.launchContext()->getCuDnnHandle());
//     auto res = hipdnnSetStream(*handle, *block.launchContext()->getCudaStream());
//     if (res != 0)
//         throw nd4j::cuda_exception::build("Can't set stream for cuDNN", res);

//     auto input   = INPUT_VARIABLE(0);                                    // [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
//     auto weights = INPUT_VARIABLE(1);                                    // [kH, kW, iC, oC] always
//     auto bias    = block.width() > 2 ? INPUT_VARIABLE(2) : nullptr;      // [oC]

//     auto output  = OUTPUT_VARIABLE(0);                                   // [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

//     NDArray::prepareSpecialUse({output}, {input, weights, bias});

//     int sH = INT_ARG(2);                                                        // strides height
//     int sW = INT_ARG(3);                                                        // strides width
//     int pH = INT_ARG(4);                                                        // paddings height
//     int pW = INT_ARG(5);                                                        // paddings width
//     int dH = INT_ARG(6);                                                        // dilations height
//     int dW = INT_ARG(7);                                                        // dilations width
//     int isSameMode = INT_ARG(8);                                                // 0-VALID, 1-SAME
//     bool isNCHW    = block.getIArguments()->size() > 9 ? !INT_ARG(9) : 1;       // INT_ARG(9): 0-NCHW,  1-NHWC

//     int kH = INT_ARG(0) > 0 ? INT_ARG(0) : static_cast<int>(weights->sizeAt(0)); // filter(kernel) height
//     int kW = INT_ARG(1) > 0 ? INT_ARG(1) : static_cast<int>(weights->sizeAt(1)); // filter(kernel) width

//     int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
//     int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
//     ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);
//     ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW, isSameMode);

//     auto dtype = cudnnDataType(input->dataType());


//     hipdnnTensorDescriptor_t src;
//     hipdnnCreateTensorDescriptor(&src);
//     res = hipdnnSetTensor4dDescriptorEx(src, dtype, input->sizeAt(0), input->sizeAt(1), input->sizeAt(2), input->sizeAt(3), input->strideAt(0), input->strideAt(1), input->strideAt(2), input->strideAt(3));
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx src failed", res);

//     // TODO: we definitely want NHWC here as well
//     hipdnnFilterDescriptor_t wght;
//     hipdnnCreateFilterDescriptor(&wght);
//     res = hipdnnSetFilter4dDescriptor(wght, dtype, HIPDNN_TENSOR_NCHW, oC, iC, kH, kW);
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnSetFilter4dDescriptor failed", res);

//     hipdnnConvolutionDescriptor_t cdc;
//     hipdnnCreateConvolutionDescriptor(&cdc);
//     res = hipdnnSetConvolution2dDescriptor(cdc, pH, pW, sH, sW, dH, dW, HIPDNN_CROSS_CORRELATION, dtype);
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnSetConvolution2dDescriptor failed", res);

//     hipdnnTensorDescriptor_t dst;
//     hipdnnCreateTensorDescriptor(&dst);
//     res = hipdnnSetTensor4dDescriptorEx(dst, dtype, output->sizeAt(0), output->sizeAt(1), output->sizeAt(2), output->sizeAt(3), output->strideAt(0), output->strideAt(1), output->strideAt(2), output->strideAt(3));
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx dst failed", res);

//     // TODO: workspace algorithms are supposed to be faster, so we should use it here if we have enough memory
//     hipdnnConvolutionFwdAlgo_t algo;
//     res = hipdnnGetConvolutionForwardAlgorithm(*handle, src, wght, cdc, dst, HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &algo);
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnGetConvolutionForwardAlgorithm failed", res);

//     // TODO: should be float if dtype is half/float, and double otherwise
//     float alpha = 1.0f;
//     float beta = 0.0f;
//     res = hipdnnConvolutionForward(*handle, &alpha, src, input->specialBuffer(), wght, weights->specialBuffer(), cdc, algo, nullptr, 0, &beta, dst, output->specialBuffer());
//     if (res != 0)
//         throw nd4j::cuda_exception::build("hipdnnConvolutionForward failed", res);


//     if (bias != nullptr) {
//         hipdnnTensorDescriptor_t bs;
//         hipdnnCreateTensorDescriptor(&bs);
//         if (isNCHW) {
//             res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NCHW, dtype, 1, bias->lengthOf(), 1, 1);
//             if (res != 0)
//                 throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
//         } else {
//             res = hipdnnSetTensor4dDescriptor(bs, HIPDNN_TENSOR_NHWC, dtype, 1, 1, 1, bias->lengthOf());
//             if (res != 0)
//                 throw nd4j::cuda_exception::build("hipdnnSetTensor4dDescriptorEx bias NHWC failed", res);
//         }

//         res = hipdnnAddTensor(*handle, &alpha, bs, bias->specialBuffer(), &alpha, dst, output->specialBuffer());
//         if (res != 0)
//             throw nd4j::cuda_exception::build("hipdnnAddTensor failed", res);
//     }


//     NDArray::registerSpecialUse({output}, {input, weights, bias});

//     return Status::OK();
// }

PLATFORM_CHECK(conv2d, ENGINE_CUDA) {
    return true;
}



}
}
}
